#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/common.hpp"
#include "caffe/blob.hpp"
#include "caffe/layers/hough_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void InitHoughBasis(const int H_, const int W_,
                               const int THETA_, const int RHO_,
                               const Dtype* sin_, const Dtype* cos_,
                               const int rho_min_, const Dtype rho_step_,
                               Dtype* val_, int* ro_, int* ci_) {
  CUDA_KERNEL_LOOP(idx, H_*W_*THETA_) {
    const int hw = idx / THETA_;
    const int theta_i = idx % THETA_;
    const int h = hw / W_;
    const int w = hw % W_;
    const int ro = hw * THETA_;

    Dtype rho = h*sin_[theta_i] + w*cos_[theta_i];
    int rho_i = int( (rho-rho_min_)/rho_step_ );
    int ci = theta_i * RHO_ + rho_i;  // col idx
    val_[ro+theta_i] = Dtype(1);
    ci_[ro+theta_i] = ci;

    if (theta_i == 0) {
      ro_[hw] = ro;
      if (idx == H_*W_*THETA_-1) {
        ro_[hw+1] = ro + THETA_;
      }
    }
  }
}

template <typename Dtype>
void HoughBasis<Dtype>::Init_gpu() {
  const Dtype pi = std::acos(-1);
  SyncedMemory theta_(sizeof(Dtype) * THETA_);
  for (int theta_i = 0; theta_i < THETA_; theta_i++) {
    Dtype theta = theta_min_ + theta_i * theta_step_;
    ((Dtype*) theta_.mutable_cpu_data())[theta_i] = theta * pi / 180;
  }
  SyncedMemory sin_(sizeof(Dtype) * THETA_);
  SyncedMemory cos_(sizeof(Dtype) * THETA_);
  caffe_gpu_sincos(THETA_, (const Dtype*) theta_.gpu_data(),
                   (Dtype*) sin_.mutable_gpu_data(),
                   (Dtype*) cos_.mutable_gpu_data());
  InitHoughBasis<<<CAFFE_GET_BLOCKS(H_*W_*THETA_), CAFFE_CUDA_NUM_THREADS>>>(
      H_, W_, THETA_, RHO_, (const Dtype*) sin_.gpu_data(), (const Dtype*) cos_.gpu_data(),
      rho_min_, rho_step_, csr_val_mutable_gpu_data(), csr_ro_mutable_gpu_data(),
      csr_ci_mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
  caffe_gpu_csr2csc(H_*W_, THETA_*RHO_, H_*W_*THETA_,
                    csr_val_gpu_data(), csr_ro_gpu_data(), csr_ci_gpu_data(),
                    csc_val_mutable_gpu_data(), csc_ri_mutable_gpu_data(),
                    csc_co_mutable_gpu_data());
}

template void HoughBasis<float>::Init_gpu();
template void HoughBasis<double>::Init_gpu();

/**
 * HoughLayer
 */

template <typename Dtype>
void HoughLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                    const vector<Blob<Dtype>*>& top) {
  caffe_gpu_csrmv(CblasNoTrans, hb_ptr_->RHO()*hb_ptr_->THETA(),
                  hb_ptr_->H()*hb_ptr_->W(), hb_ptr_->nnz(),
                  Dtype(1), hb_ptr_->csc_val_gpu_data(),
                  hb_ptr_->csc_co_gpu_data(), hb_ptr_->csc_ri_gpu_data(),
                  bottom[0]->gpu_data(), Dtype(0), top[0]->mutable_gpu_data());
}

template <typename Dtype>
void HoughLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                     const vector<bool>& propagate_down,
                                     const vector<Blob<Dtype>*>& bottom) {
  caffe_gpu_csrmv(CblasNoTrans, hb_ptr_->H()*hb_ptr_->W(),
                  hb_ptr_->RHO()*hb_ptr_->THETA(), hb_ptr_->nnz(),
                  Dtype(1), hb_ptr_->csr_val_gpu_data(),
                  hb_ptr_->csr_ro_gpu_data(), hb_ptr_->csr_ci_gpu_data(),
                  top[0]->gpu_diff(), Dtype(0), bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(HoughLayer);

/**
 * HoughTransposeLayer
 */

template <typename Dtype>
void HoughTransposeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                             const vector<Blob<Dtype>*>& top) {
  caffe_gpu_csrmv(CblasNoTrans, hb_ptr_->H()*hb_ptr_->W(),
                  hb_ptr_->RHO()*hb_ptr_->THETA(), hb_ptr_->nnz(),
                  Dtype(1), hb_ptr_->csr_val_gpu_data(),
                  hb_ptr_->csr_ro_gpu_data(), hb_ptr_->csr_ci_gpu_data(),
                  bottom[0]->gpu_data(), Dtype(0), top[0]->mutable_gpu_data());
}

template <typename Dtype>
void HoughTransposeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                              const vector<bool>& propagate_down,
                                              const vector<Blob<Dtype>*>& bottom) {
  caffe_gpu_csrmv(CblasNoTrans, hb_ptr_->RHO()*hb_ptr_->THETA(),
                  hb_ptr_->H()*hb_ptr_->W(), hb_ptr_->nnz(),
                  Dtype(1), hb_ptr_->csc_val_gpu_data(),
                  hb_ptr_->csc_co_gpu_data(), hb_ptr_->csc_ri_gpu_data(),
                  top[0]->gpu_diff(), Dtype(0), bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(HoughTransposeLayer);

}  // namespace caffe
